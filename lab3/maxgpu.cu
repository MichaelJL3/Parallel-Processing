#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <ctime>

using namespace std;

//assignment constraints prevent the optimization of this function
//better approach would have been to take a max per block
//then sort the resulting maxes, with blocks of thread size 1024 you can cut down the serial
//sort from array size to (array size)/1024 or 2^(n-10) where n=log2(array size) assuming it is a power of 2

__global__ void getmaxcu(const unsigned int* numI, unsigned int* numO, const unsigned int size){
	extern __shared__ unsigned int sarr[];

	//thread id
	const unsigned int tdx=threadIdx.x;

	//global location in array
	unsigned int gdx=(blockDim.x)*blockIdx.x+threadIdx.x;

	//compare two initially
	sarr[tdx]=numI[gdx];

    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if (tdx<s)
            sarr[tdx]=max(sarr[tdx],sarr[tdx+s]);
        __syncthreads();
    }

	if(!tdx){
		//atomically swap output value for multiple blocks
		atomicMax(numO, sarr[0]);
	}
}

int main(int argc, char *argv[]){

	hipError_t err;
	unsigned int *numbers_temp_in, *numbers_temp_out;
	unsigned int *numbers_in, *numbers_out;
	unsigned int size;

	//blocksize is 1024 based on cuda5 (original test source) and then my own GPU after all the servers crashed
	const unsigned int blockSize=1024;

	//check for proper args
    if(argc!=2){
       printf("usage: maxseq num\nnum = size of the array\n");
       exit(1);
    }
   
	//get the size and then determine number of blocks in grid
    size = atol(argv[1]);  
	const unsigned int gridSize=ceil((float)size/blockSize);

	//numbers_out = single max element to return
	numbers_out=(unsigned int *)malloc(sizeof(unsigned int));
	if(!numbers_out){
		printf("Failed Allocation\n");
		exit(1);
	} 

	//numbers_in = array of randomized inputs based on size
	numbers_in=(unsigned int *)malloc(size*sizeof(unsigned int));
	if(!numbers_in){
		printf("Failed Allocation\n");
		exit(1);
	} 

	//input the array
    srand(time(NULL)); // setting a seed for the random number generator
    for(unsigned int i=0; i<size; ++i)
       numbers_in[i] = rand()%size;  

	//allocate the memory in the GPU
	hipMalloc((void**)&numbers_temp_in, sizeof(unsigned int)*size);
	hipMalloc((void**)&numbers_temp_out, sizeof(unsigned int));

	//copt over the inputs to the device
    hipMemcpy((void*)numbers_temp_in, (void*)numbers_in, size*sizeof(unsigned int), hipMemcpyHostToDevice);

	//set the grid/block dimensions
	dim3 dimGrid(gridSize);
	dim3 dimBlock(blockSize);

	//run the gpu max
	getmaxcu<<<dimGrid,dimBlock,blockSize*sizeof(unsigned int)>>>(numbers_temp_in, numbers_temp_out, size);

	//print out some information
	cout<<"Starting GPU Calculation:\nBlocksize: "<<blockSize<<"\nBlocks in grid: "<<gridSize<<"\n";

	//wait until completion
	hipDeviceSynchronize();
	err=hipGetLastError();
	if(err!=hipSuccess){
		printf("Err: %s\n", hipGetErrorString(err));
		exit(1);
	}

	//get the output back from the device
	hipMemcpy((void*)numbers_out, (void*)numbers_temp_out, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	err=hipGetLastError();
	if(err!=hipSuccess){
		printf("Err Getting Output: %s\n", hipGetErrorString(err));
		exit(1);
	}

	//print the max value	
	cout<<"The maximum number in the array is: "<<*numbers_out<<"\n";

	//free memory
	hipFree(numbers_temp_in);
	hipFree(numbers_temp_out);
	free(numbers_in);
	free(numbers_out);

	return 0;
}
